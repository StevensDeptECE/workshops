#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, const float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main() {
  int N = 1<<20; // 1024*1024 = 1M
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float)); // allocate memory on CPU
  y = (float*)malloc(N*sizeof(float));
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMalloc(&d_x, N*sizeof(float));  // d_x = new allocated memory on GPU
  hipMalloc(&d_y, N*sizeof(float));
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice); //copy to GPU
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost); // copy back to CPU

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x); // free the memory on the GPU
  hipFree(d_y);
  free(x);  // free the memory on the CPU
  free(y);
}